#include "hip/hip_runtime.h"
// Tellusim header

#include <common/common.h>
#include <common/sample_controls.h>
#include <platform/TellusimDevice.h>
#include <platform/TellusimContext.h>
#include <platform/TellusimPipeline.h>
#include <platform/TellusimCommand.h>

/*
 */
__global__ void kernel(uint32_t size, float scale, float time, float4 *positions) {
	
	uint32_t global_x = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t global_y = blockDim.y * blockIdx.y + threadIdx.y;
	
	uint32_t id = global_y * size + global_x;
	
	float x = (float)global_x / size * 2.0f - 1.0f;
	float y = (float)global_y / size * 2.0f - 1.0f;
	
	float r = sin(x * scale) * 0.5f + 0.5f;
	float g = cos(y * scale) * 0.5f + 0.5f;
	float b = max(1.0f - r - g, 0.0f);
	
	uint32_t color = 0xff000000u;
	color |= (uint32_t)(r * 255.0f) << 0u;
	color |= (uint32_t)(g * 255.0f) << 8u;
	color |= (uint32_t)(b * 255.0f) << 16u;
	
	positions[id] = make_float4(x * scale, y * scale, r + g + sin(sqrt(x * x + y * y) * 4.0f + time * 2.0f) * 4.0f, __uint_as_float(color));
}

/*
 */
int32_t main(int32_t argc, char **argv) {
	
	using namespace Tellusim;
	
	DECLARE_WINDOW
	
	// create window
	String title = String::format("%s Tellusim::Cuda Runtime", window.getPlatformName());
	if(!window.create(title) || !window.setHidden(false)) return 1;
	
	// geometry parameters
	constexpr uint32_t grid_size = 1024;
	constexpr uint32_t group_size = 8;
	
	// structures
	struct CommonParameters {
		Matrix4x4f projection;
		Matrix4x4f modelview;
	};
	
	// create device
	Device device(window);
	if(!device) return 1;
	
	// create Cuda context for our device
	CUContext cu_context = CUContext(Context(PlatformCU, device.getFeatures().pciBusID));
	if(!cu_context.create()) {
		TS_LOG(Error, "main(): can't create Cuda context\n");
		return 1;
	}
	
	// create Cuda device
	Device cu_device(cu_context);
	if(!cu_device) return 1;
	
	// set Cuda device
	if(hipSetDevice(cu_context.getDevice()) != hipSuccess) return 1;
	
	// Cuda info
	int32_t driver_version = 0;
	int32_t runtime_version = 0;
	if(hipDriverGetVersion(&driver_version) != hipSuccess) return 1;
	if(hipRuntimeGetVersion(&runtime_version) != hipSuccess) return 1;
	TS_LOGF(Message, "Driver: %u\n", driver_version);
	TS_LOGF(Message, "Runtime: %u\n", runtime_version);
	
	// create pipeline
	Pipeline pipeline = device.createPipeline();
	pipeline.setUniformMask(0, Shader::MaskVertex);
	pipeline.addAttribute(Pipeline::AttributePosition, FormatRGBAf32, 0, 0, sizeof(float32_t) * 4);
	pipeline.setColorFormat(window.getColorFormat());
	pipeline.setDepthFormat(window.getDepthFormat());
	pipeline.setPrimitive(Pipeline::PrimitivePoint);
	pipeline.setDepthFunc(Pipeline::DepthFuncLessEqual);
	if(!pipeline.loadShaderGLSL(Shader::TypeVertex, "main.shader", "VERTEX_SHADER=1")) return 1;
	if(!pipeline.loadShaderGLSL(Shader::TypeFragment, "main.shader", "FRAGMENT_SHADER=1")) return 1;
	if(!pipeline.create()) return 1;
	
	// create position buffer
	Buffer position_buffer = device.createBuffer(Buffer::FlagStorage | Buffer::FlagVertex | Buffer::FlagInterop, sizeof(float32_t) * 4 * grid_size * grid_size);
	if(!position_buffer) return 1;
	
	// create Cuda position buffer
	CUBuffer cu_position_buffer = CUBuffer(cu_device.createBuffer(position_buffer));
	if(!cu_position_buffer) return 1;
	
	// create target
	Target target = device.createTarget(window);
	
	// create canvas
	Canvas canvas;
	
	// create panel
	ControlRoot root(canvas, true);
	ControlPanel panel(&root, 1, 8.0f, 8.0f);
	panel.setAlign(Control::AlignRight | Control::AlignTop);
	panel.setPosition(-8.0f, -8.0f);
	
	// create sliders
	ControlSlider scale_slider(&panel, "Scale", 3, 32.0f, 16.0f, 48.0f);
	scale_slider.setSize(192.0f, 0.0f);
	
	// main loop
	DECLARE_GLOBAL
	window.run([&]() -> bool {
		DECLARE_COMMON
		
		// suppress warnings
		simulate = simulate;
		pause = pause;
		
		Window::update();
		
		if(!window.render()) return false;
		
		// window title
		if(fps > 0.0f) window.setTitle(String::format("%s %.1f FPS", title.get(), fps));
		
		// update controls
		update_controls(window, root);
		canvas.create(device, target);
		
		// dispatch Cuda kernel
		{
			// dispatch Cuda kernel
			uint32_t num_groups = udiv(grid_size, group_size);
			hipStream_t stream = (hipStream_t)cu_context.getStream();
			float4 *data = (float4*)cu_position_buffer.getBufferPtr();
			kernel<<<dim3(num_groups, num_groups), dim3(8, 8), 0, stream>>>(grid_size, scale_slider.getValuef32(), time, data);
			
			// check Cuda error
			hipError_t error = hipGetLastError();
			if(error != hipSuccess) TS_LOGF(Error, "main(): %s\n", hipGetErrorString(error));
			
			// synchronize stream
			hipStreamSynchronize(stream);
		}
		
		// flush buffer
		device.flushBuffer(position_buffer);
		
		// window target
		target.setClearColor(0.2f, 0.2f, 0.2f, 1.0f);
		target.begin();
		{
			// create command list
			Command command = device.createCommand(target);
			
			// set pipeline
			command.setPipeline(pipeline);
			
			// set position buffers
			command.setVertexBuffer(0, position_buffer);
			
			// set common parameters
			CommonParameters common_parameters;
			common_parameters.projection = Matrix4x4f::perspective(60.0f, (float32_t)window.getWidth() / window.getHeight(), 0.1f, 1000.0f);
			common_parameters.modelview = Matrix4x4f::lookAt(Vector3f(20.0f, 20.0f, 20.0f), Vector3f(0.0f, 0.0f, 0.0f), Vector3f(0.0f, 0.0f, 1.0f));
			if(target.isFlipped()) common_parameters.projection = Matrix4x4f::scale(1.0f, -1.0f, 1.0f) * common_parameters.projection;
			command.setUniform(0, common_parameters);
			
			// draw geometry
			command.drawArrays(grid_size * grid_size);
			
			// draw canvas
			canvas.draw(command, target);
		}
		target.end();
		
		if(!window.present()) return false;
		
		if(!device.check()) return false;
		
		return true;
	});
	
	// finish context
	window.finish();
	
	return 0;
}
